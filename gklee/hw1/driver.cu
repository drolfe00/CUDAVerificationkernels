
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
//include the header file for your library here
//#include "cudafunctions.cu"

#define BLOCKSIZE  32

__global__ void d_add( float *x, float *y, float *z, int size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < size)
		z[index] = x[index] + y[index];
}

void add( float *x, float *y, int length)
{

	float *d_x, *d_y, *d_z;  // device copies of x and y and a result z

	int size = length * sizeof(float);  // need space for total number of floats

	// allocate device space
	hipMalloc( (void**)&d_x, size);	 
	hipMalloc( (void**)&d_y, size);	 
	hipMalloc( (void**)&d_z, size);

	// copy vector from host to device
	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);	 
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	// launch the kernel, eat some chicken
	d_add<<< ceil((float)length/(float)BLOCKSIZE), BLOCKSIZE >>>(d_x, d_y, d_z, size);

	// copy the result back to the host
	hipMemcpy(x, d_z, size, hipMemcpyDeviceToHost);

	// free device mem
	hipFree(d_x);	 
	hipFree(d_y);	 
	hipFree(d_z);

	// hope for the best	 
}



/**
* mul:
* cuda vector multiply function
**/


__global__ void d_mul( float *x, float *y, float *z, int size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < size)
		z[index] = x[index] * y[index];

}


void mul( float *x, float *y, int length)
{

	float *d_x, *d_y, *d_z;  // device copies of x and y and a result z

	int size = length * sizeof(float);  // need space for total number of floats

	// allocate device space
	hipMalloc( (void**)&d_x, size);	 
	hipMalloc( (void**)&d_y, size);	 
	hipMalloc( (void**)&d_z, size);

	// copy vector from host to device
	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);	 
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	// launch the kernel, eat some chicken
	d_mul<<< ceil((float)length/(float)BLOCKSIZE), BLOCKSIZE >>>(d_x, d_y, d_z, size);

	// copy the result back to the host
	hipMemcpy(x, d_z, size, hipMemcpyDeviceToHost);

	// free device mem
	hipFree(d_x);	 
	hipFree(d_y);	 
	hipFree(d_z);

}






int func_add ( float *x, float *y, int sz)
{
	int i;
	float *a;
	a = ( float *)malloc(sizeof(float)*sz);
	if (!a){
		printf("memory allocation error\n");
		exit(-1);
	}
	memcpy(a,x,sz*(sizeof(float)));

	/* replace the code to add
         * with a cuda call which you will
	 * implement as a interface to your cuda enabled library
	 */
	/*
	for ( i=0; i<sz; i++)
		x[i]+=y[i];
	*/
	// replace with cuda enabled call
	add(x, y, sz);
		
	for ( i=0; i<sz; i++){
		if (x[i]!= a[i] + y[i]){
			printf("x = %f, a = %f, y = %f, i = %d, size = %d  ", x[i], a[i], y[i], i, sz);
			return 0;
			}
	}
		
	free(a);
	return 1;
}

	 	
int func_mul ( float *x, float *y, int sz)
{
	int i;
	float *a;
	a = ( float *)malloc(sizeof(float)*sz);
	if (!a){
		printf("memory allocation error\n");
		exit(-1);
	}
	memcpy(a,x,sz*(sizeof(float)));

	/* replace the code to multiply
         * with a cuda call which you will
	 * implement as a interface to your cuda enabled library
	 */
	/*
	for ( i=0; i<sz; i++)
		x[i]*=y[i];
	*/
	// cuda call
	mul(x, y, sz);	
	for ( i=0; i<sz; i++){
		if (x[i]!= a[i] * y[i]){
			printf("x = %f, a = %f, y = %f, i = %d, size = %d  ", x[i], a[i], y[i], i, sz);
			return 0;
			}
	}
	
	free(a);
	return 1;
}

int main()
{
	
	float *a,*b;
	int j;
	int i;

	for ( j=10; j<1000000; j*=10){
		a =( float *) malloc(sizeof(float)*j);
		b =( float *) malloc(sizeof(float)*j);

		for (i=0; i<j; i++){
			a[i] = 2;
			b[i] = 3;
		}

		if(!func_add(a,b,j)){
			printf("failed to add\n");
			}
		else{
			printf("add operation completed\n");
			}
		
		if(!func_mul(a,b,j)){
			printf("failed to mul\n");
			}
		else{
			printf("mul operation completed\n");
			}	
		
		free(a);
		free(b);
	}

		return 0;
}
